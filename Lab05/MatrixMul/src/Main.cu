#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

#include <hip/hip_runtime.h>
#include <hipblaslt.h>

#include "Functions.h"
#include "Matrix.h"

#define matrixType float
#define BLOCK_SIZE 16 //256 threads per block

int main(int argc, char **argv)
{
	int widthA = 0;
	int heightA = 0;
	int widthB = 0;
	int heightB = 0;
	std::cout<<"Enter MatrixA width = MatrixB height"<<std::endl;
	std::cin >> widthA;
	heightB = widthA;
	std::cout<<"Enter MatrixB width"<<std::endl;
	std::cin >> widthB;

	Matrix matA = Matrix(widthA, heightA, cuda);
	Matrix matB = Matrix(widthB, heightB, cuda);
	InitializeMatrix(matA);
	InitializeMatrix(matB);

	int resWidth = matB._width;
	int resHeight = matA._height;

	Matrix matCNaive = Matrix(resWidth, resHeight, cuda);
	Matrix matCShared = Matrix(resWidth, resHeight, cuda);
	Matrix matCCublas = Matrix(resWidth, resHeight, cuda);

	Matrix matCCPU = Matrix(resWidth, resHeight);

	//Compute grid sizes
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 dimGrid((resWidth + dimBlock.x - 1)/dimBlock.x,(resHeight + dimBlock.x - 1)/dimBlock.y);

	//Launch Kernels
	double ti = cpuTimer();
	MatrixMulShared<<<dimGrid, dimBlock>>>(matA, matB, matCShared);
	hipDeviceSynchronize();
	double sharedTime = cpuTimer() - ti;
	ti = cpuTimer();
	MatrixMulNaive<<<dimGrid, dimBlock>>>(matA , matB, matCNaive);
	hipDeviceSynchronize();
	double naiveTime = cpuTimer() - ti;
	//Multiply using cuBLAS
	ti = cpuTimer();
	CublasMultiply(matA, matB, matCCublas);
	double cublasTime = cpuTimer() - ti;
	//Check results on CPU
	MatrixMul(matA, matB, matCCPU);

	std::cout<<"Results:"<<std::endl
			<<"Naive: "<<naiveTime<<std::endl
			<<"Shared: "<<sharedTime<<std::endl
			<<"cuBLAS: "<<cublasTime<<std::endl<<std::endl;
	std::cout<<"Checking shared memory version"<<std::endl;
	MatrixCompare(matCCPU, matCShared);
	std::cout<<"Checking naive version"<<std::endl;
	MatrixCompare(matCCPU, matCNaive);
	std::cout<<"Checking cuBLAS version"<<std::endl;
	MatrixCompare(matCCPU, matCCublas);
}
