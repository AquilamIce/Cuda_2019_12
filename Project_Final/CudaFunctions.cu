#include "hip/hip_runtime.h"
#include "CudaFunctions.h" // Header file
#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "Matrix.h" // Header file of Matrix class


//Matrix addition kernel
//Pointers to array of elements of Matrix A, Matrix B and Matrix C. Values of Matrix A size.
__global__ void MatrixAdd(const float *A_elements, const float *B_elements,  float *C_elements, const int A_width, const int A_height)
{

	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	if (row < A_height && col < A_width)
	{
	 	//Modifying array of elements of Matrix C
		C_elements[row * A_width + col] = A_elements[row * A_width + col] + B_elements[row * A_width + col];
	}
}

//Matrix substraction kernel
//Pointers to array of elements of Matrix A, Matrix B and Matrix C. Values of Matrix A size.
__global__ void MatrixSubtract(const float* A_elements, const float* B_elements,  float* C_elements, const int A_width, const int A_height)
{
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	if (row < A_height && col < A_width)
	{
	 	//Modifying array of elements of Matrix C
		C_elements[row * A_width + col] = A_elements[row * A_width + col] - B_elements[row * A_width + col];
	}
}

//Matrix multiplication kernel
//Pointers to array ofelements of Matrix A, Matrix B and Matrix C.
//Values of Matrix A, Matrix B and Matrix C size.
__global__ void MatrixMultiply(const float* A_elements, const float* B_elements,  float* C_elements, const int A_width, const int A_height, const int B_width, const int B_height, int C_width,  int C_height)
{
	 int Row = blockIdx.y * blockDim.y + threadIdx.y;
	 int Col = blockIdx.x * blockDim.x + threadIdx.x;

     __shared__ float As[TILE_SIZE][TILE_SIZE];
     __shared__ float Bs[TILE_SIZE][TILE_SIZE];

    float Cvalue = 0;

    int rowInBlock = threadIdx.y;
    int colInBlock = threadIdx.x;


    for (int i = 0; i < ((A_width + TILE_SIZE - 1) / TILE_SIZE); ++i)
    {
    	if((Row < A_height) && ((colInBlock + i*TILE_SIZE) < A_width))
    		As[rowInBlock][colInBlock] = A_elements[Row*A_width + colInBlock + i*TILE_SIZE];
    	else
    		As[rowInBlock][colInBlock] = 0;

    	if((Col < B_width) && ((rowInBlock + i*TILE_SIZE) < B_height))
    		Bs[rowInBlock][colInBlock] = B_elements[(rowInBlock + i*TILE_SIZE)*B_width + Col];
    	else
			Bs[rowInBlock][colInBlock] = 0;

        //Synchronize threads
        __syncthreads();

        for (int j = 0; j < TILE_SIZE; ++j)
        {
        		Cvalue += As[rowInBlock][j] * Bs[j][colInBlock];
        }

        __syncthreads();
    }

    if (Row < C_height && Col < C_width) //Saving Final result into Matrix C
    {
        C_elements[Row*C_width + Col] = Cvalue;
    }
}

//Matrix transposition kernel
//Pointers to array of elements of Matrix A and Matrix B. Values of Matrix A height and Matrix B width.
__global__ void MatrixTranspose(const float *A_elements, float *B_elements, const int A_width, const int A_height, const int B_width)
{
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	if (row < A_width && col < A_height)
	{
		B_elements[row * B_width + col] = A_elements[col * A_width + row];
	}
}

//Matrix elements summation kernel
//Forwarded Matrix input and recived pointer to output value.
__global__ void VectorSum(Matrix input, double *output)
{
	__shared__ double partialSum[2*BLOCK_SIZE];

	unsigned int thread = threadIdx.x;
	unsigned int start = 2 * blockIdx.x * BLOCK_SIZE;
	//Each thread copy 2 elements from input into shared memory
	if (start + thread < input._width)
		partialSum[thread] = input._elements[start+thread];
	else
		partialSum[thread] = 0;
	if (start + BLOCK_SIZE + thread < input._width)
		partialSum[BLOCK_SIZE + thread] = input._elements[start + BLOCK_SIZE + thread];
	else
		partialSum[BLOCK_SIZE + thread] = 0;

	//Use reduction to calculate sum of each block
	for (int stride{BLOCK_SIZE}; stride >= 1; stride >>= 1)
	{
		__syncthreads();
		if (thread < stride)
			partialSum[thread] += partialSum[thread + stride];
	}

	//Save sum of each block into output
	if (thread == 0)
	{
		//Atomic function guarantees that threads will not try to write into memory at the same time
		atomicAdd(output, partialSum[0]);
	}
}
