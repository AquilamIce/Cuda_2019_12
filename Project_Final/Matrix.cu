#include "hip/hip_runtime.h"
#include "Matrix.h"
#include <stdlib.h>
#include <iostream>
#include <hip/hip_runtime.h>
#include <stdio.h>


Matrix::Matrix(int width, int height)
{
	_width = width;
	_height = height;

	hipMallocManaged(&_elements, width * height * sizeof(float));
}

Matrix::Matrix(const Matrix &matrix)
{
	_width = matrix._width;
	_height = matrix._height;
	hipMallocManaged(&_elements, _width * _height * sizeof(float));

	for(int i = 0; i < matrix._width * matrix._height; i++)
	{
		_elements[i] = matrix._elements[i];
	}
}

Matrix::~Matrix()
{
	hipFree(_elements);
}

void Matrix::Initialize(float num, bool randomize)
{
	if(randomize)
	{
		for(int i = 0; i < _width * _height; i++)
		{
			_elements[i] = rand()/(float)RAND_MAX;;
		}
	}
	else
	{
		for(int i = 0; i < _width * _height; i++)
		{
			_elements[i] = num;
		}
	}
}

void Matrix::Print()
{
	for(int row = 0; row < _height; row ++)
	{
		for(int col = 0; col < _width; col++)
		{
			printf("M[%d;%d] = %f\t", row + 1, col + 1, _elements[(row*_width) + col]);
		}
		printf("\n");
	}
}

void Matrix::MatrixCompare(Matrix &A, Matrix &B)
{
	for(int row = 0; row < A._height; row++)
	{	for(int col = 0; col < A._width; col++)
		{
			if(fabs(A._elements[(row*A._width) + col] - B._elements[(row*B._width) + col]) > 1e-4)
			{
				fprintf(stderr, "Result verification failed at element M[%d;%d]!\t%f|%f\n", row+1, col+1, A._elements[(row*A._width) + col], B._elements[(row*B._width) + col]);
			}
		}
	}
	std::cout<<"Check ok"<<std::endl;
}


Matrix Matrix::operator+(const Matrix &matrix)
{
	if (matrix._width != _width || matrix._height != _height)
	{
		fprintf(stderr, "Wrong matrix size");
		exit(EXIT_FAILURE);
	}

	Matrix result = Matrix(_width, _height);

	 for (int i = 0; i < _width * _height; i++)
	 {
	 	result._elements[i] = this->_elements[i] + matrix._elements[i];
	 }

	return result;
}

Matrix Matrix::operator-(const Matrix &matrix)
{
	if (matrix._width != _width || matrix._height != _height)
	{
		fprintf(stderr, "Wrong matrix size");
		exit(EXIT_FAILURE);
	}

	Matrix result = Matrix(_width, _height);

	for (int i = 0; i < _width * _height; i++)
	{
		result._elements[i] = this->_elements[i] - matrix._elements[i];
	}

	return result;
}

Matrix Matrix::operator*(const Matrix &matrix)
{
	if (_width != matrix._height)
	{
		fprintf(stderr, "wrong matrix size");
		exit(EXIT_FAILURE);
	}

	Matrix result = Matrix(matrix._width, _height);

	for(int row = 0; row < result._height; row++)
	{
		for(int col = 0; col < result._width; col++)
		{
			for(int k = 0; k < _width; k++)
				result._elements[(row*result._width) + col] += _elements[(row*_width) + k] * matrix._elements[col + (k*matrix._width)];
		}
	}

	return result;
}

const Matrix& Matrix::operator=(const Matrix &A)
{
  if (&A == this)
    return *this;
  hipFree(_elements);
  _width = A._width;
  _height = A._height;

  hipMallocManaged(&_elements, _width * _height * sizeof(float));

  for(int i = 0; i < A._width * A._height; ++i)
  {
	  this->_elements[i] = A._elements[i];
  }

  return *this;
}

Matrix Matrix::Transpose()
{
	Matrix result = Matrix(_height, _width);
	for (int row = 0; row < result._height; row++)
		for (int col = 0; col < result._width; col++)
			result._elements[row*result._width + col] = _elements[col*_width + row];
	return result;
}

float Matrix::VectorSum()
{
	float result = 0;

	for(int i = 0; i < _width * _height; i++)
	{
		result += _elements[i];
	}
	return result;
}

float& Matrix::operator[](int i)
{
	return _elements[i];
}

const float& Matrix::operator[](int i) const
{
	return _elements[i];
}
