#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <iostream>
#include <cmath>
//#include <cudaProfiler.h>
#include <cstdio>
#include <sys/time.h>


//#define SIZE (1024)
#define BLOCK_SIZE 16

__global__ void VectorAdd(float *VecA, float *VecB, float *VecC, int size);
void InitializeVector(float *VecA, int size);

int main(int argc, char *argv[])
{

	int SIZE = atoi(argv[1]);

	struct timeval start, end;
	gettimeofday(&start, NULL);

	//Input
	float *VecA, *VecB, *VecE, *VecG;
	//Output
	float *VecC, *VecD,  *VecF;
	hipMallocManaged(&VecA, SIZE * sizeof(float));
	hipMallocManaged(&VecB, SIZE * sizeof(float));
	hipMallocManaged(&VecC, SIZE * sizeof(float));
	hipMallocManaged(&VecD, SIZE * sizeof(float));
	hipMallocManaged(&VecE, SIZE * sizeof(float));
	hipMallocManaged(&VecF, SIZE * sizeof(float));
	hipMallocManaged(&VecG, SIZE * sizeof(float));

	//Initialize input
	InitializeVector(VecA, SIZE);
	InitializeVector(VecB, SIZE);
	InitializeVector(VecE, SIZE);
	InitializeVector(VecG, SIZE);

	//Calculate grid dimensions
	dim3 dimBlock = BLOCK_SIZE;
	dim3 dimGrid((SIZE + BLOCK_SIZE)/(BLOCK_SIZE));

	//Launch kernels
	VectorAdd<<<dimGrid, dimBlock>>>(VecA, VecB, VecC, SIZE);
	hipDeviceSynchronize();
	VectorAdd<<<dimGrid, dimBlock>>>(VecC, VecE, VecD, SIZE);
	hipDeviceSynchronize();
	VectorAdd<<<dimGrid, dimBlock>>>(VecD, VecG, VecF, SIZE);
	hipDeviceSynchronize();

	gettimeofday(&end, NULL);
	double diff = ((end.tv_sec * 1000000 + end.tv_usec) - (start.tv_sec * 1000000 + start.tv_usec));

	std::cout<<"Timing [us]: "<<diff<<std::endl;


	//Check results
    for (int i = 0; i < SIZE; i++)
    {
        if (VecA[i] + VecB[i] + VecE[i] + VecG[i] - VecF[i]  > 1e-5)
        {
            fprintf(stderr, "Result verification failed at element %d!\n", i);
            exit(EXIT_FAILURE);
        }
    }
    printf("Test passed\n");

    //Free memory
    hipFree(VecA);
    hipFree(VecB);
    hipFree(VecC);
    hipFree(VecD);
    hipFree(VecE);
    hipFree(VecF);
    hipFree(VecG);

}

__global__ void VectorAdd(float *VecA, float *VecB, float *VecC, int size)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i < size)
		VecC[i] = VecA[i] + VecB[i];
}
void InitializeVector(float *VecA, int size)
{
	for (int i = 0; i < size; i++)
		VecA[i] = rand()/(float)RAND_MAX;
}
