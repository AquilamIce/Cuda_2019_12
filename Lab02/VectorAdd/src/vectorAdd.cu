#include "hip/hip_runtime.h"
/**
 * Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

/**
 * Vector addition: C = A + B.
 *
 * This sample is a very basic sample that implements element by element
 * vector addition. It is the same as the sample illustrating Chapter 2
 * of the programming guide with some additions like error checking.
 */

#include <stdio.h>

// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>

#include <hip/hip_runtime_api.h>
/**
 * CUDA Kernel Device code
 *
 * Computes the vector addition of A and B into C. The 3 vectors have the same
 * number of elements numElements.
 */
__global__ void
vectorAdd(const float *A, const float *B, float *C, int numElements)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < numElements)
    {
        C[i] = A[i] + B[i];
    }
}

/**
 * Host main routine
 */
int
main(void)
{

	int deviceCount = 0;

    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;

    // Print the vector length to be used, and compute its size
    unsigned long numElements = 2<<29;
    size_t size = numElements * sizeof(float);
    printf("%f\n", size);
    printf("[Vector addition of %d elements]\n", numElements);

    // Allocate the vector A
    float *A;
    hipMallocManaged(&A, size);

    // Allocate the vector B
    float *B;
    hipMallocManaged(&B, size);

    // Allocate the vector C
    float *C;
    hipMallocManaged(&C, size);

    // Verify that allocations succeeded
    if (A == NULL || B == NULL || C == NULL)
    {
        fprintf(stderr, "Failed to allocate vectors!\n");
        exit(EXIT_FAILURE);
    }

    // Initialize the host input vectors
    for (int i = 0; i < numElements; ++i)
    {
        A[i] = rand()/(float)RAND_MAX;
        B[i] = rand()/(float)RAND_MAX;
    }
    // Launch the Vector Add CUDA Kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;
    printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(A, B, C, numElements);
    err = hipGetLastError();
    hipDeviceSynchronize();

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Verify that the result vector is correct
    for (int i = 0; i < numElements; ++i)
    {
        if (fabs(A[i] + B[i] - C[i]) > 1e-5)
        {
            fprintf(stderr, "Result verification failed at element %d!\n", i);
            exit(EXIT_FAILURE);
        }
    }

    printf("Test PASSED\n");

    // Free memory
    hipFree(A);
    hipFree(B);
    hipFree(C);

    printf("Done\n");
    return 0;
}

